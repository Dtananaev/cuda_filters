#include "hip/hip_runtime.h"
/*
* cuda_nlm.cpp
*
*  Created on: March 17, 2015
*      Author: Denis Tananaev
*/

#include "cuda_nlm.h"
#include <cmath>
#include <cstdio>
#include "CMatrix.h"
#include <ctime>

using namespace std;


//Cuda_NLM_Naive
__global__ void cuda_NLM_filter(float* result, const float* input,float *gauss_lut, int x_size, int y_size,int window_radius,int patch_radius,float inv_sqr_sigma)
{
int x = blockDim.x * blockIdx.x + threadIdx.x;
int y = blockDim.y * blockIdx.y + threadIdx.y;

if(x>=x_size || y>=y_size)
return;

  //variables
 int gauss_lut_center = patch_radius;
	float sum = 0; 
    float new_value = 0;
	
	

	// window
    const int x1 = max(0,x-window_radius);
    const int y1 = max(0,y-window_radius);
    const int x2 = min(x_size-1,x+window_radius);
    const int y2 = min(y_size-1,y+window_radius);

	
	//patch comparing
    for( int ny = y1; ny <= y2; ++ny )
    for( int nx = x1; nx <= x2; ++nx )
    {
		float dist = 0;
  for( int ty = -patch_radius; ty <= patch_radius; ++ty )
  for( int tx = -patch_radius; tx <= patch_radius; ++tx )
  {
    // clamp coordinates
    int p1x = min(x_size-1,max(0,x+tx));
    int p1y = min(y_size-1,max(0,y+ty));
    int p2x = min(x_size-1,max(0,nx+tx));
    int p2y = min(y_size-1,max(0,ny+ty));

	//calculate distance between patches
    float tmp = input[p1y*x_size+p1x]-input[p2y*x_size+p2x];
    dist += tmp*tmp*gauss_lut[gauss_lut_center+tx]*gauss_lut[gauss_lut_center+ty];
  }

  //calculate weight of each patch
      float w = exp(-dist*inv_sqr_sigma);
  //weightet sum of the patches
      new_value += w*input[ny*x_size+nx];
  //normalizer
      sum+= w;
    }
	//synchronaise threads
	__syncthreads();

    result[y*x_size+x] = new_value/sum;

}

CMatrix<float> cuda_NLM_Naive(const CMatrix<float> &image, int window_radius, int patch_radius, float sqr_sigma){

 CMatrix<float> result(image.xSize(), image.ySize(), 0);
 float inv_sqr_sigma = 1. / sqr_sigma;


 //gauss lut
  float* gauss_lut = new float[2*patch_radius+1];
  float* gauss_lut_center = gauss_lut+patch_radius;
  for( int i = -patch_radius; i <= patch_radius; ++i )
    *(gauss_lut_center+i) = std::exp(-0.5*i*i/(patch_radius*patch_radius));

  //memory allocation of the device
  float* d_image;
  float* d_gauss_lut;
  float* d_result;

  hipMalloc((void**)&d_image, image.size()*sizeof(float));
  hipMalloc((void**)&d_result, image.size()*sizeof(float));
   hipMalloc((void**)&d_gauss_lut, (2*patch_radius+1)*sizeof(float));

  // copy image to device
  hipMemcpy(d_image, image.data(), 
              image.size()*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy( d_gauss_lut, gauss_lut, 
             (2*patch_radius+1)*sizeof(float), hipMemcpyHostToDevice);

  //kernel
  dim3 block(32, 32, 1);
  dim3 grid;
  grid.x = std::ceil(image.xSize()/(float)block.x);
  grid.y = std::ceil(image.ySize()/(float)block.y);
//naive filter
     cuda_NLM_filter<<<grid,block>>>(d_result, d_image, d_gauss_lut, image.xSize(), image.ySize(),window_radius,patch_radius, inv_sqr_sigma);


  // Copy result back
  hipMemcpy( (void*)result.data(), (void*)d_result, 
              result.size()*sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_image);
	hipFree(d_result);
	hipFree(d_gauss_lut);
  return result;
}
